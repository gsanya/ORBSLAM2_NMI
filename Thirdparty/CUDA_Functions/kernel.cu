/**
* This file is part of orbslam2_NMI.
*
* Copyright (C) 2021 S�ndor Gazdag <gazdag.sandor at sztaki dot hu> (SZTAKI)
* For more information see <https://github.com/gsanya/orbslam2_NMI>
*
* orbslam2_NMI is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* orbslam2_NMI is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with ORB-SLAM2. If not, see <http://www.gnu.org/licenses/>.
*/
#pragma once
#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <>
#include <hip/hip_runtime_api.h>

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <chrono>
#include <stdio.h>

#include <opencv2/features2d/features2d.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>


#include "NMI.cuh"
#include "kernel.cuh"




namespace CUDAF {
	void NMIWithCuda_noMask(cv::cuda::PtrStep<unsigned char> *d_data2GPU, int NMI_mode,  int MatchingMode, int width, int height, float *NMI, unsigned int syntGL)
	{
		//Map openGL texture to cuda array		
		hipGraphicsResource_t resources[1] = {0};
		checkCudaErrors(hipGraphicsGLRegisterImage(&resources[0], syntGL, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly));
		checkCudaErrors(hipGraphicsMapResources(1, resources, 0));
		hipArray *synthCUDA;
		checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&synthCUDA, resources[0], 0, 0));
		
		//GPU memory is with d_(device) prefix, CPU memory is with h_(host) prefix
		uint  *d_Histogram1, *d_Histogram2, *d_JointHistogram;
		float * d_Entropy1, *d_Entropy2, *d_JointEntropy, *d_JointEntropyShort;

		//for nice look
		dim3 blocksPerGrid(0, 0, 0);
		dim3 threadsPerBlock(0, 0, 0);
		//asd

		checkCudaErrors(hipMalloc((void **)&d_Histogram1, 256 * sizeof(uint)));
		checkCudaErrors(hipMalloc((void **)&d_Histogram2, 256 * sizeof(uint)));
		checkCudaErrors(hipMalloc((void **)&d_JointHistogram, 256 * 256 * sizeof(uint)));
		checkCudaErrors(hipMalloc((void **)&d_Entropy1, 256 * sizeof(float)));
		checkCudaErrors(hipMalloc((void **)&d_Entropy2, 256 * sizeof(float)));
		checkCudaErrors(hipMalloc((void **)&d_JointEntropyShort, 256 * sizeof(float)));
		checkCudaErrors(hipMalloc((void **)&d_JointEntropy, 256 * 256 * sizeof(float)));
		

		initHistogram256all();

		//calculates histogram and joint histogram
		histogram256all(d_JointHistogram, d_Histogram1, d_Histogram2, (uchar*)d_data2GPU, width, height, synthCUDA);													
	

		//computes "entropy" for each histogram bin
		blocksPerGrid = { 258, 1, 1 };
		threadsPerBlock = { 256, 1, 1 }; 
		ComputeEntropyKernel << < blocksPerGrid, threadsPerBlock >> > (d_Histogram1, d_Histogram2, d_JointHistogram, width*height, d_Entropy1, d_Entropy2, d_JointEntropy);	


		blocksPerGrid = { 256,1,1 };
		threadsPerBlock = { 128,1,1 };
		AddvectorParwiseMidKernel << < blocksPerGrid, threadsPerBlock >> > (d_JointEntropy, d_JointEntropyShort);

		//calculates the entropies, and than the nmi to d_Entropy1[0]
		blocksPerGrid = { 3,1,1 };
		threadsPerBlock = { 128,1,1 };
		AddVectorPairwiseKernel << < blocksPerGrid, threadsPerBlock >> > (d_Entropy1, d_Entropy2, d_JointEntropyShort);
					
		closeHistogram256all();				

		//copy the calculated NMI to the CPU
		checkCudaErrors(hipMemcpy((void*)&NMI[0], d_Entropy1,  sizeof(float), hipMemcpyDeviceToHost));

		//deletes
		checkCudaErrors(hipFree(d_Histogram1));
		checkCudaErrors(hipFree(d_Histogram2));
		checkCudaErrors(hipFree(d_JointHistogram));
		checkCudaErrors(hipFree(d_Entropy1));
		checkCudaErrors(hipFree(d_Entropy2));
		checkCudaErrors(hipFree(d_JointEntropyShort));
		checkCudaErrors(hipFree(d_JointEntropy));

		//unmap openGL resources
		checkCudaErrors(hipGraphicsUnmapResources(1,resources));
		checkCudaErrors(hipGraphicsUnregisterResource(resources[0]));
	}
}
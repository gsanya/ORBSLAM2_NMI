#include "hip/hip_runtime.h"
/**
* This file is part of orbslam2_NMI.
*
* Copyright (C) 2021 S�ndor Gazdag <gazdag.sandor at sztaki dot hu> (SZTAKI)
* For more information see <https://github.com/gsanya/orbslam2_NMI>
*
* orbslam2_NMI is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* orbslam2_NMI is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with ORB-SLAM2. If not, see <http://www.gnu.org/licenses/>.
*/

#pragma once
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>

#include <opencv2/core/core.hpp>
#include <opencv2/features2d/features2d.hpp>
#include <opencv2/cudawarping.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "NMI.cuh"
#include "kernel.cuh"
#include "allProperties.hpp"

//texture for opengl rendered texture
texture<uchar, hipTextureType2D, hipReadModeElementType> texCUDA;

//atomics
inline __device__ void addByte_noBG(uint tid, uint *d_PartialJointHistograms, uint *s_WarpHist1, uint *s_WarpHist2, uint data1, uint data2)
{
	uint d1 = data1;
	uint d2 = data2;
	atomicAdd(s_WarpHist1 + d1, 1); //atomicAdd(memloc, value to add)
	atomicAdd(s_WarpHist2 + d2, 1);
	atomicAdd(d_PartialJointHistograms + (tid >> LOG2_WARP_SIZE) * JOINT_HISTOGRAM256_BIN_COUNT + d1 * HISTOGRAM256_BIN_COUNT + d2, 1);
}


__global__ void histogram256Kernel(
	uint *d_PartialJointHistograms,
	uint *d_PartialHistograms1,
	uint *d_PartialHistograms2,
	uchar *d_Warped,
	uint width,
	uint height)
{
	//shared histograms (one for every warp)
	__shared__ uint s_Hist1[HISTOGRAM256_THREADBLOCK_MEMORY]; //  warp_count * 256 = 16*256 = 4096
	__shared__ uint s_Hist2[HISTOGRAM256_THREADBLOCK_MEMORY]; //  warp_count * 256 --> �sszesen 8192*4= 32768 Byte= 32 kByte shared memory k�ne blokkonk�nt (nvidia Geforce 950M-nek 65536 Byte, szval el�g)


	// calculating starting position for the warp
	uint *s_WarpHist1 = s_Hist1 + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM256_BIN_COUNT;
	uint *s_WarpHist2 = s_Hist2 + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM256_BIN_COUNT;


	// Clear shared memory storage for current threadblock before processing
#pragma unroll
	for (uint i = 0; i < (HISTOGRAM256_THREADBLOCK_MEMORY / HISTOGRAM256_THREADBLOCK_SIZE); i++)
	{
		s_Hist1[threadIdx.x + i * HISTOGRAM256_THREADBLOCK_SIZE] = 0;
		s_Hist2[threadIdx.x + i * HISTOGRAM256_THREADBLOCK_SIZE] = 0;
	}
	__syncthreads();	//syncs threads in block

	for (uint pos = UMAD(blockIdx.x, blockDim.x, threadIdx.x); pos < width*height; pos += UMUL(blockDim.x, gridDim.x))
	{

		uint data1 = uint(tex2D(texCUDA, float(pos % width), float(height - 1) - float(pos / width)));
		uint data2 = uint(d_Warped[pos]);

		if (nmi_prop_BG || ((data1 != 0) && (data2 != 0)))
			addByte_noBG(threadIdx.x, d_PartialJointHistograms, s_WarpHist1, s_WarpHist2, data1, data2);
	}
	__syncthreads();

	//Merge per-warp histograms into per-block and write to global memory
	for (uint bin = threadIdx.x; bin < HISTOGRAM256_BIN_COUNT; bin += HISTOGRAM256_THREADBLOCK_SIZE)
	{
		uint sum1 = 0;
		uint sum2 = 0;
		for (uint i = 0; i < WARP_COUNT256; i++)
		{
			sum1 += s_Hist1[bin + i * HISTOGRAM256_BIN_COUNT];
			sum2 += s_Hist2[bin + i * HISTOGRAM256_BIN_COUNT];
		}
		// per block sub-histogram 
		d_PartialHistograms1[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] = sum1;
		d_PartialHistograms2[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] = sum2;
	}
}


#define MERGE_THREADBLOCK_SIZE 1024


__global__ void mergeHistogram256Kernel(
	uint *d_Histogram1,
	uint *d_Histogram2,
	uint *d_PartialHistograms1,
	uint *d_PartialHistograms2,
	uint histogramCount)
{
	uint sum1 = 0;
	uint sum2 = 0;
	

	for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)	
	{
		sum1 += d_PartialHistograms1[blockIdx.x + i * HISTOGRAM256_BIN_COUNT];
		sum2 += d_PartialHistograms2[blockIdx.x + i * HISTOGRAM256_BIN_COUNT];
	}

	//shared only inside blocks
	__shared__ uint data1[MERGE_THREADBLOCK_SIZE];
	__shared__ uint data2[MERGE_THREADBLOCK_SIZE];

	data1[threadIdx.x] = sum1;
	data2[threadIdx.x] = sum2;
	for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
	{
		__syncthreads();
		if (threadIdx.x < stride)
		{
			data1[threadIdx.x] += data1[threadIdx.x + stride];
			data2[threadIdx.x] += data2[threadIdx.x + stride];
		}
	}
	if (threadIdx.x == 0)
	{
		d_Histogram1[blockIdx.x] = data1[0];
		d_Histogram2[blockIdx.x] = data2[0];
	}
}

__global__ void mergeJointHistogram256Kernel(
	uint *d_JointHistogram,
	uint *d_PartialHistograms,
	uint jointHistogramCount)
{
	uint sum = 0;
#pragma unroll
	for (int i = 0; i < jointHistogramCount; i++)
	{
		sum += d_PartialHistograms[blockIdx.x*blockDim.x + threadIdx.x + i * JOINT_HISTOGRAM256_BIN_COUNT];
	}
	d_JointHistogram[blockIdx.x*blockDim.x + threadIdx.x] = sum;
}


static const uint  PARTIAL_HISTOGRAM_COUNT = 240;
static uint        *d_PartialHistograms1;
static uint        *d_PartialHistograms2;
static uint        *d_PartialJointHistograms;


//Internal memory allocation
extern "C" void initHistogram256all(void)
{
	checkCudaErrors(hipMalloc((void **)&d_PartialHistograms1, PARTIAL_HISTOGRAM_COUNT * HISTOGRAM256_BIN_COUNT * sizeof(uint)));
	checkCudaErrors(hipMalloc((void **)&d_PartialHistograms2, PARTIAL_HISTOGRAM_COUNT * HISTOGRAM256_BIN_COUNT * sizeof(uint)));
	checkCudaErrors(hipMalloc((void **)&d_PartialJointHistograms, WARP_COUNT256 * JOINT_HISTOGRAM256_BIN_COUNT * sizeof(uint)));

}

//Internal memory deallocation
extern "C" void closeHistogram256all(void)
{
	checkCudaErrors(hipFree(d_PartialHistograms1));
	checkCudaErrors(hipFree(d_PartialHistograms2));
	checkCudaErrors(hipFree(d_PartialJointHistograms));
}

// wrapper function
extern "C" void histogram256all(
	uint *d_JointHistogram,
	uint *d_Histogram1,
	uint *d_Histogram2,
	uchar *d_Warped,
	uint width,
	uint height,
	hipArray *synthCUDA
)
{
	checkCudaErrors(hipMemset(d_PartialJointHistograms, 0, WARP_COUNT256 * JOINT_HISTOGRAM256_BIN_COUNT * sizeof(uint)));

	checkCudaErrors(hipBindTextureToArray(texCUDA, synthCUDA));

	//<<<240,512>>>
	histogram256Kernel << <PARTIAL_HISTOGRAM_COUNT, HISTOGRAM256_THREADBLOCK_SIZE >> > (
		d_PartialJointHistograms,
		d_PartialHistograms1,
		d_PartialHistograms2,
		d_Warped,
		width,
		height);

	//<<<256,1024>>>
	mergeHistogram256Kernel << <HISTOGRAM256_BIN_COUNT, MERGE_THREADBLOCK_SIZE >> > (
		d_Histogram1,
		d_Histogram2,
		d_PartialHistograms1,
		d_PartialHistograms2,
		PARTIAL_HISTOGRAM_COUNT);

	// <<< 256 , 256 >>>
	mergeJointHistogram256Kernel << < HISTOGRAM256_BIN_COUNT, HISTOGRAM256_BIN_COUNT >> > (
		d_JointHistogram,
		d_PartialJointHistograms,
		WARP_COUNT256);

	checkCudaErrors(hipUnbindTexture(texCUDA));
}


//<<<258,256>>
__global__ void ComputeEntropyKernel(
	uint *d_Histogram1,
	uint* d_Histogram2,
	uint *d_JointHistogram,
	int length,
	float* d_EntropyArray1,
	float* d_EntropyArray2,
	float* d_JointEntropyArray)
{

	if (blockIdx.x == 0)
	{
		if (d_Histogram1[threadIdx.x] == 0)
			d_EntropyArray1[threadIdx.x] = 0;
		else
			d_EntropyArray1[threadIdx.x] = ((float)d_Histogram1[threadIdx.x] / (float)length)		*		log2f((float)d_Histogram1[threadIdx.x] / (float)length);
	}
	else
	{
		if (blockIdx.x == 1)
		{
			if (d_Histogram2[threadIdx.x] == 0)
				d_EntropyArray2[threadIdx.x] = 0;
			else
				d_EntropyArray2[threadIdx.x] = ((float)d_Histogram2[threadIdx.x] / (float)length)		*		log2f((float)d_Histogram2[threadIdx.x] / (float)length);
		}
		else
		{
			if (d_JointHistogram[blockDim.x*(blockIdx.x - 2) + threadIdx.x] == 0)
				d_JointEntropyArray[blockDim.x*(blockIdx.x - 2) + threadIdx.x] = 0;
			else
			{
				int ind = blockDim.x*(blockIdx.x - 2) + threadIdx.x;
				d_JointEntropyArray[ind] = ((float)d_JointHistogram[ind] / (float)length)		 *			log2f((float)d_JointHistogram[ind] / (float)length);
			}
		}
	}
}

//<<<256, 128>>>
__global__ void AddvectorParwiseMidKernel(
	float* d_Array,
	float *d_out)
{
	int t = blockDim.x*blockIdx.x * 2 + threadIdx.x;
	int n = blockDim.x;
	while (n >= 1)
	{
		if (t - blockDim.x*blockIdx.x * 2 < n)
		{
			d_Array[t] += d_Array[t + n];
		}
		__syncthreads();
		n /= 2;
	}
	if (threadIdx.x == 0)
		d_out[blockIdx.x] = d_Array[blockDim.x*blockIdx.x * 2];
}


__global__ void AddVectorPairwiseKernel(
	float* d_Array1,
	float* d_Array2,
	float* d_Array3)
{
	if (blockIdx.x == 0)//summing Ha
	{
		int t = threadIdx.x;
		int n = blockDim.x;
		while (n != 0)
		{
			if (t < n)
			{
				d_Array1[t] += d_Array1[t + n];
			}
			__syncthreads();
			n /= 2;
		}
	}
	else
	{
		if (blockIdx.x == 1)//summing Hb
		{
			int t = threadIdx.x;
			int n = blockDim.x;
			while (n != 0)
			{
				if (t < n)
				{
					d_Array2[t] += d_Array2[t + n];
				}
				__syncthreads();
				n /= 2;
			}
		}
		else//summing Hab
		{
			int t = threadIdx.x;
			int n = blockDim.x;
			while (n != 0)
			{
				if (t < n)
				{
					d_Array3[t] += d_Array3[t + n];
				}
				__syncthreads();
				n /= 2;
			}
		}
	}
	//__syncthreads();
	//calculate NMI into d_Array1[0] on the first thread
	if (blockIdx.x == 1 && threadIdx.x == 0)
	{
		if (ENMI) {
			if (d_Array1[0] == 0 && d_Array2[0] == 0 && d_Array3[0] == 0) {
				d_Array1[0] = 0;
			}
			else {
				d_Array1[0] = ((-d_Array1[0])+(-d_Array2[0]))/(-d_Array3[0]);
			}			
		}
		else if(SUC){	
			if (d_Array1[0] == 0 && d_Array2[0] == 0 && d_Array3[0] == 0) {
				d_Array1[0] = 0;
			}
			else {
				d_Array1[0] = 2*(1-((-d_Array3[0])/((-d_Array1[0]) + (-d_Array2[0]))));
			}
		}
		else
			d_Array1[0] = -1;
	}
}



